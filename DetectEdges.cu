
// Allow creating vectors of filenames of training images and encodings.
#include <vector>

// Allow creating elements of vectors.
#include <string>

// Allow use of GetFilenamesOfTrainingImages.
#include "GetFilenamesOfTrainingImages.h"

// Allow use of GetFilenamesOfEncodingsForTrainingImages.
#include "GetFilenamesOfEncodingsForTrainingImages.h"

// Allow use of GetInputTensorOnHost.
#include "GetInputTensorOnHost.h"

// Allow
// - creating cudnnHandle,
// - creating inputTensorDescriptor,
// - creating filterTensorDescriptor,
#include <hipDNN.h>

#include "DisplayImagesInActivationOutputTensor.h"

// --------------------------------------
// Allows use of std::cout and std::endl.
#include <iostream>
// --------------------------------------

int main()
{
	// -------------------------------------------------------------
	// Define vectors of filenames of training images and encodings.
	// -------------------------------------------------------------
	// Define a path to training images and encodings.
	std::string pathToImages = "Images\\";

	// Define a vector of filenames of training images.
	std::vector<std::string> filenamesOfTrainingImages = GetFilenamesOfTrainingImages(pathToImages);

	// -------------------
	// Create cudnnHandle.
	// -------------------
	hipdnnHandle_t cudnnHandle;
	hipdnnCreate(&cudnnHandle);

	// ------------------------
	// Define inputTensorOnGPU.
	// ------------------------
	// Define properties of inputTensor.
	int imagesInSubdivision = 4;
	int channelsInImage = 3;
	int heightOfImage = 416;
	int widthOfImage = 416;
	int elementsInImage = channelsInImage * heightOfImage * widthOfImage;
	int elementsInInputTensor = imagesInSubdivision * channelsInImage * heightOfImage * widthOfImage;

	// Define inputTensorDescriptor.
	hipdnnTensorDescriptor_t inputTensorDescriptor;
	hipdnnCreateTensorDescriptor(&inputTensorDescriptor);
	hipdnnSetTensor4dDescriptor(
		/*tensorDesc=*/inputTensorDescriptor,
		/*format=*/HIPDNN_TENSOR_NCHW,
		/*dataType=*/HIPDNN_DATA_DOUBLE,
		/*n=*/imagesInSubdivision,
		/*c=*/channelsInImage,
		/*h=*/heightOfImage,
		/*w=*/widthOfImage);

	// Declare a pointer for training images in host heap that will be used by GetInputTensorOnHost.
	// I must use a pointer to access image buffer.
	// I encounter a runtime error if I define and delete the pointer in GetInputTensorOnHost.
	double* pointerToTrainingImage = new double[elementsInImage];

	// Define a pointer to inputTensorOnHost in host heap.
	double* inputTensorOnHost = new double[elementsInInputTensor];
	GetInputTensorOnHost(
		inputTensorOnHost,
		pointerToTrainingImage,
		imagesInSubdivision,
		channelsInImage,
		heightOfImage,
		widthOfImage,
		pathToImages,
		filenamesOfTrainingImages);

	// Delete pointer for training images.
	delete pointerToTrainingImage;

	// Define a pointer to inputTensorOnGPU in GPU heap.
	double* inputTensorOnGPU;
	hipMalloc(&inputTensorOnGPU, elementsInInputTensor * sizeof(double));
	hipMemcpy(inputTensorOnGPU, inputTensorOnHost, elementsInInputTensor * sizeof(double), hipMemcpyHostToDevice);

	// Delete pointer inputTensorOnHost.
	delete inputTensorOnHost;

	// -------------------------
	// Define filterTensorOnGPU.
	// -------------------------
	// Define properties of filterTensor.
	const int filters = 3;
	const int channelsInFilter = 3;
	const int heightOfFilter = 3;
	const int widthOfFilter = 3;
	const int elementsInFilterTensor = filters * channelsInFilter * heightOfFilter * widthOfFilter;

	// Define filterTensorDescriptor.
	hipdnnFilterDescriptor_t filterTensorDescriptor;
	hipdnnCreateFilterDescriptor(&filterTensorDescriptor);
	hipdnnSetFilter4dDescriptor(
		/*filterDesc=*/filterTensorDescriptor,
		/*dataType=*/HIPDNN_DATA_DOUBLE,
		/*format=*/HIPDNN_TENSOR_NCHW,
		/*k=*/filters,
		/*c=*/channelsInFilter,
		/*h=*/heightOfFilter,
		/*w=*/widthOfFilter
	);

	// Define pointer to filterTensorOnHost in host heap.
	double* filterTensorOnHost = new double[elementsInFilterTensor];
	double channelTemplate[3][3] = {
		{1.0,1.0,1.0},
		{1.0,-8.0,1.0},
		{1.0,1.0,1.0}
	};
	for (int f = 0; f < filters; ++f)
	{
		for (int c = 0; c < channelsInFilter; ++c)
		{
			for (int h = 0; h < heightOfFilter; ++h)
			{
				for (int w = 0; w < widthOfFilter; ++w)
				{
					filterTensorOnHost[w + h*widthOfFilter + c*heightOfFilter*widthOfFilter + f*channelsInFilter*heightOfFilter*widthOfFilter] =
						channelTemplate[h][w];
				}
			}
		}
	}

	// Define pointer to filterTensorOnGPU in GPU heap.
	double* filterTensorOnGPU;
	hipMalloc(&filterTensorOnGPU, elementsInFilterTensor * sizeof(double));
	hipMemcpy(filterTensorOnGPU, filterTensorOnHost, elementsInFilterTensor * sizeof(double), hipMemcpyHostToDevice);

	// Delete pointer filterTensorOnHost.
	delete filterTensorOnHost;

	// --------------------
	// Perform convolution.
	// --------------------
	// Define convolutionHyperparametersDescriptor.
	hipdnnConvolutionDescriptor_t convolutionHyperparametersDescriptor;
	hipdnnCreateConvolutionDescriptor(&convolutionHyperparametersDescriptor);
	hipdnnSetConvolution2dDescriptor(
		/*convDesc=*/convolutionHyperparametersDescriptor,
		/*pad_h=*/1,
		/*pad_w=*/1,
		/*u= vertical filter stride =*/1,
		/*v= horizontal filter stride =*/1,
		/*dilation_h=*/1,
		/*dilation_w=*/1,
		/*mode=*/HIPDNN_CROSS_CORRELATION,
		/*computeType=*/HIPDNN_DATA_DOUBLE);

	// Define properties of outputTensor.
	int outputSubtensors = 0;
	int channelsInOutputSubtensor = 0;
	int heightOfOutputSubtensor = 0;
	int widthOfOutputSubtensor = 0;

	hipdnnGetConvolution2dForwardOutputDim(
		/*convDesc=*/convolutionHyperparametersDescriptor,
		/*inputTensorDesc=*/inputTensorDescriptor,
		/*filterDesc=*/filterTensorDescriptor,
		/**n=*/&outputSubtensors,
		/**c=*/&channelsInOutputSubtensor,
		/**h=*/&heightOfOutputSubtensor,
		/**w*/&widthOfOutputSubtensor);

	int elementsInOutputTensor = outputSubtensors * channelsInOutputSubtensor * heightOfOutputSubtensor * widthOfOutputSubtensor;

	// Create outputTensorDescriptor.
	hipdnnTensorDescriptor_t outputTensorDescriptor;
	hipdnnCreateTensorDescriptor(&outputTensorDescriptor);
	hipdnnSetTensor4dDescriptor(
		/*tensorDesc=*/outputTensorDescriptor,
		/*format=*/HIPDNN_TENSOR_NCHW,
		/*datatType=*/HIPDNN_DATA_DOUBLE,
		/*n=*/outputSubtensors,
		/*c=*/channelsInOutputSubtensor,
		/*h=*/heightOfOutputSubtensor,
		/*w=*/widthOfOutputSubtensor);

	// Define convolutionAlgorithmDescriptor.
	// After call to hipdnnGetConvolutionForwardAlgorithm,
	// convolutionAlgorithm = 0 (HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM).
	hipdnnConvolutionFwdAlgo_t convolutionAlgorithmDescriptor;
	hipdnnGetConvolutionForwardAlgorithm(
		/*handle=*/cudnnHandle,
		/*xDesc=*/inputTensorDescriptor,
		/*wDesc=*/filterTensorDescriptor,
		/*convDesc=*/convolutionHyperparametersDescriptor,
		/*yDesc=*/outputTensorDescriptor,
		/*preference=*/HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
		/*memoryLimitInBytes=*/0, // No limit.
		/**algo=*/&convolutionAlgorithmDescriptor);

	// Declare pointer to workspaceOnGPU in GPU heap.
	size_t bytesInWorkspace;

	hipdnnGetConvolutionForwardWorkspaceSize(
		/*handle=*/cudnnHandle,
		/*xDesc=*/inputTensorDescriptor,
		/*wDesc=*/filterTensorDescriptor,
		/*convDesc=*/convolutionHyperparametersDescriptor,
		/*yDesc=*/outputTensorDescriptor,
		/*algo=*/convolutionAlgorithmDescriptor,
		/**sizeInBytes=*/&bytesInWorkspace);

	void* workspaceOnGPU;
	hipMalloc(&workspaceOnGPU, bytesInWorkspace);

	// Define pointer to convolutionOutputTensorOnGPU in GPU heap.
	double* convolutionOutputTensorOnGPU;
	hipMalloc(&convolutionOutputTensorOnGPU, elementsInOutputTensor * sizeof(double));

	// Perform convolution.
	double one = 1.0;
	double zero = 0.0;

	hipdnnConvolutionForward(
		/*handle=*/cudnnHandle,
		/**alpha=*/&one,
		/*xDesc=*/inputTensorDescriptor,
		/**x=*/inputTensorOnGPU,
		/*wDesc=*/filterTensorDescriptor,
		/*w=*/filterTensorOnGPU,
		/*convDesc=*/convolutionHyperparametersDescriptor,
		/*algo=*/convolutionAlgorithmDescriptor,
		/*workSpace=*/workspaceOnGPU,
		/*workSpaceSizeInBytes=*/bytesInWorkspace,
		/**beta=*/&zero,
		/*yDesc=*/outputTensorDescriptor,
		/**y=*/convolutionOutputTensorOnGPU);

	// Free inputTensorOnGPU.
	hipFree(inputTensorOnGPU);

	// Free filterTensorOnGPU.
	hipFree(filterTensorOnGPU);

	// Free workspaceOnGPU.
	hipFree(workspaceOnGPU);

	// ----------------------------
	// Perform batch normalization.
	// ----------------------------
	// Define pointer to batchNormOutputTensorOnGPU in GPU heap.
	double* batchNormOutputTensorOnGPU;
	hipMalloc(&batchNormOutputTensorOnGPU, elementsInOutputTensor * sizeof(double));

	// Define batchNormMode.
	hipdnnBatchNormMode_t batchNormMode = HIPDNN_BATCHNORM_SPATIAL;

	// DeclareDefine derivedBNTensorDescriptor.
	hipdnnTensorDescriptor_t derivedBNTensorDescriptor;
	hipdnnCreateTensorDescriptor(&derivedBNTensorDescriptor);
	hipdnnDeriveBNTensorDescriptor(
		/*derivedBNDesc=*/derivedBNTensorDescriptor,
		/*xDesc=*/outputTensorDescriptor,
		/*mode=*/batchNormMode);

	// Define pointer to batchNormScalesOnGPU in GPU heap.
	double* batchNormScalesOnHost = new double[channelsInOutputSubtensor];
	int c;
	for (c = 0; c < channelsInOutputSubtensor; ++c) {
		batchNormScalesOnHost[c] = 1.0;
	}
	double* batchNormScalesOnGPU;
	hipMalloc(&batchNormScalesOnGPU, channelsInOutputSubtensor * sizeof(double));
	hipMemcpy(batchNormScalesOnGPU, batchNormScalesOnHost, channelsInOutputSubtensor * sizeof(double), hipMemcpyHostToDevice);
	delete batchNormScalesOnHost;

	// Define pointer to batchNormBiasesOnGPU in GPU heap.
	double* batchNormBiasesOnHost = new double[channelsInOutputSubtensor];
	for (c = 0; c < channelsInOutputSubtensor; ++c) {
		batchNormBiasesOnHost[c] = 0.0;
	}
	double* batchNormBiasesOnGPU;
	hipMalloc(&batchNormBiasesOnGPU, channelsInOutputSubtensor * sizeof(double));
	hipMemcpy(batchNormBiasesOnGPU, batchNormBiasesOnHost, channelsInOutputSubtensor * sizeof(double), hipMemcpyHostToDevice);
	delete batchNormBiasesOnHost;

	// Define Exponential Average Factor. 
	double expAverageFactor = 1.0;

	// Define pointer to resultRunningMeansOnGPU in GPU heap.
	double* resultRunningMeansOnGPU;
	hipMalloc(&resultRunningMeansOnGPU, channelsInOutputSubtensor * sizeof(double));

	// Define pointer to resultRunningVariancesOnGPU in GPU heap.
	double* resultRunningVariancesOnGPU;
	hipMalloc(&resultRunningVariancesOnGPU, channelsInOutputSubtensor * sizeof(double));

	// Define epsilon.
	double epsln = 0.00001;

	// Define pointer to resultSaveMeanOnGPU in GPU heap.
	double* resultSaveMeanOnGPU;
	hipMalloc(&resultSaveMeanOnGPU, channelsInOutputSubtensor * sizeof(double));

	// Define pointer to resultSaveInvVarianceOnGPU in GPU heap.
	double* resultSaveInvVarianceOnGPU;
	hipMalloc(&resultSaveInvVarianceOnGPU, channelsInOutputSubtensor * sizeof(double));

	hipdnnBatchNormalizationForwardTraining(
		/*handle=*/cudnnHandle,
		/*mode=*/batchNormMode,
		/**alpha=*/&one,
		/**beta=*/&zero,
		/*xDesc=*/outputTensorDescriptor,
		/**x=*/convolutionOutputTensorOnGPU,
		/*yDesc=*/outputTensorDescriptor,
		/**y=*/batchNormOutputTensorOnGPU,
		/*bnScaleBiasMeanVarDesc=*/derivedBNTensorDescriptor,
		/*bnScaleData=*/batchNormScalesOnGPU,
		/*bnBiasData=*/batchNormBiasesOnGPU,
		/*exponentialAverageFactor=*/expAverageFactor,
		/*resultRunningMeanData=*/resultRunningMeansOnGPU,
		/*resultRunningVarianceData=*/resultRunningVariancesOnGPU,
		/*epsilon=*/epsln,
		/*resultSaveMean=*/resultSaveMeanOnGPU,
		/*resultSaveInvVariance=*/resultSaveInvVarianceOnGPU);

	// Free convolutionOutputTensorOnGPU.
	hipFree(convolutionOutputTensorOnGPU);

	// Free batchNormScalesOnGPU.
	hipFree(batchNormScalesOnGPU);

	// Free batchNormBiasesOnGPU.
	hipFree(batchNormBiasesOnGPU);

	// -----------------------------
	// Perform nonlinear activation.
	// -----------------------------
	// Define activationDescriptor.
	hipdnnActivationDescriptor_t activationDescriptor;
	hipdnnCreateActivationDescriptor(&activationDescriptor);
	hipdnnSetActivationDescriptor(
		/*activationDesc=*/activationDescriptor,
		/*mode=*/HIPDNN_ACTIVATION_RELU,
		/*reluNanOpt=*/HIPDNN_PROPAGATE_NAN,
		/*coef=*/std::numeric_limits<double>::infinity());

	// Define pointer to activationOutputTensorOnGPU in GPU heap.
	double* activationOutputTensorOnGPU;
	hipMalloc(&activationOutputTensorOnGPU, elementsInOutputTensor * sizeof(double));

	// Perform activation.
	hipdnnActivationForward(
		/*handle=*/cudnnHandle,
		/*activationDesc=*/activationDescriptor,
		/**alpha=*/&one,
		/**xDesc=*/outputTensorDescriptor,
		/**x=*/batchNormOutputTensorOnGPU,
		/**beta=*/&zero,
		/*yDesc=*/outputTensorDescriptor,
		/**y=*/activationOutputTensorOnGPU);

	// Free batchNormOutputTensorOnGPU.
	hipFree(batchNormOutputTensorOnGPU);

	// -------------------------------------------
	// Copy activationOutputTensor on GPU to host.
	// -------------------------------------------
	// Define pointer to activationOutputTensorOnHost in host heap.
	double* activationOutputTensorOnHost = new double[elementsInOutputTensor];
	hipMemcpy(
		activationOutputTensorOnHost,
		activationOutputTensorOnGPU,
		elementsInOutputTensor * sizeof(double),
		hipMemcpyDeviceToHost);

	hipFree(activationOutputTensorOnGPU);

	// -----------------------------------------------
	// Display images in activationOutputTensorOnHost.
	// -----------------------------------------------
	DisplayImagesInActivationOutputTensor(
		activationOutputTensorOnHost,
		outputSubtensors,
		heightOfOutputSubtensor,
		widthOfOutputSubtensor,
		channelsInOutputSubtensor);

	// Free activationOutputTensorOnHost.
	delete activationOutputTensorOnHost;

	hipFree(resultRunningMeansOnGPU);
	hipFree(resultRunningVariancesOnGPU);
	hipFree(resultSaveMeanOnGPU);
	hipFree(resultSaveInvVarianceOnGPU);

	// -----------------------------
	// Indicate that main succeeded.
	// -----------------------------
	return 0;

} // main